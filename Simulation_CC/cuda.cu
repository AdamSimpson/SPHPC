#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"

template <typename T>
__global__ void kernel(T lam) { lam(); }

int main(int argc, char **argv) {
  int *arr;
  hipMallocManaged(&arr, 1*sizeof(int));

  auto body = void { arr[0] = 7; };
  auto lambda = [=] __device__ { arr[0] = 7; };

  kernel<<<1,1>>>(lambda);
  lambda();

  hipDeviceSynchronize();
  std::cout<<arr[0]<<std::endl;

  return 0;
}
